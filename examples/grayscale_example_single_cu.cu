#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <vector>
#include "image_library/image.hpp"
#include "image_library/processing_cu.h"
#include <chrono>

// Example testing the implementation
int main(int argc, char *argv[])
{
    long long int image_size = std::pow(2048, 1);
    image_library::Image img(2048, 2048);

    // Convert the image to grayscale using the CUDA implementation
    auto start = std::chrono::high_resolution_clock::now();
    image_library::convertToGrayscaleCuda(img);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    std::cout << "Time taken: " << duration.count() << " seconds" << std::endl;

    // Output the first few pixel values to verify the grayscale conversion
    for (int i = 0; i < 5; ++i)
    {
        std::cout << "Pixel " << i << ": "
                  << static_cast<int>(img.getPixelData()[i * 3]) << ", "
                  << static_cast<int>(img.getPixelData()[i * 3 + 1]) << ", "
                  << static_cast<int>(img.getPixelData()[i * 3 + 2]) << std::endl;
    }

    return 0;
}