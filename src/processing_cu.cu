#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <vector>
#include "image_library/processing_cu.h"

namespace image_library
{
    // Kernel function for converting the image to grayscale
    __global__ void convertToGrayscaleKernel(uint8_t *pixelData, int width, int height)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < width * height)
        {
            int r = pixelData[idx * 3 + 0];
            int g = pixelData[idx * 3 + 1];
            int b = pixelData[idx * 3 + 2];
            uint8_t grayscale = static_cast<uint8_t>(0.299 * r + 0.587 * g + 0.114 * b);
            pixelData[idx * 3 + 0] = grayscale;
            pixelData[idx * 3 + 1] = grayscale;
            pixelData[idx * 3 + 2] = grayscale;
        }
    }

    // Function to handle memory allocation and kernel launch
    void convertToGrayscaleCuda(Image &img)
    {
        int width = img.getWidth();
        int height = img.getHeight();
        auto &pixelData = img.getPixelData();
        printf("Run on Cuda\n");

        uint8_t *d_pixelData;
        size_t dataSize = width * height * 3 * sizeof(uint8_t);

        // Allocate device memory
        hipMalloc(&d_pixelData, dataSize);

        // Copy pixel data to device
        hipMemcpy(d_pixelData, pixelData.data(), dataSize, hipMemcpyHostToDevice);

        // Launch kernel with sufficient threads
        int threadsPerBlock = 256;
        int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;
        convertToGrayscaleKernel<<<blocksPerGrid, threadsPerBlock>>>(d_pixelData, width, height);

        // Wait for the kernel to finish
        hipDeviceSynchronize();

        // Copy the processed pixel data back to host
        hipMemcpy(pixelData.data(), d_pixelData, dataSize, hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(d_pixelData);
    }

    // CUDA kernel for converting image to grayscale
    __global__ void convertToGrayscaleKernelParallel(uint8_t *image_data, int width, int height, int num_images)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < num_images * width * height)
        {
            int image_idx = idx / (width * height);
            int pixel_idx = idx % (width * height);
            int pixel_base = image_idx * width * height * 3 + pixel_idx * 3;

            uint8_t r = image_data[pixel_base];
            uint8_t g = image_data[pixel_base + 1];
            uint8_t b = image_data[pixel_base + 2];

            // Convert to grayscale using the average method
            uint8_t gray = static_cast<uint8_t>(0.2989f * r + 0.5870f * g + 0.1140f * b);

            // Store the grayscale value in place of the original RGB values
            image_data[pixel_base] = gray;
            image_data[pixel_base + 1] = gray;
            image_data[pixel_base + 2] = gray;
        }
    }

    // Host function to manage grayscale conversion for multiple images
    void convertToGrayscaleParallel(std::vector<image_library::Image> &images, int num_images)
    {
        int width = images[0].getWidth();
        int height = images[0].getHeight();
        size_t image_size = width * height * 3 * sizeof(uint8_t);

        // Allocate memory for image data on the device
        uint8_t *d_image_data;
        hipMalloc(&d_image_data, num_images * image_size);

        // Copy image data from host to device
        for (int i = 0; i < num_images; ++i)
        {
            hipMemcpy(d_image_data + i * width * height * 3, images[i].getPixelData().data(), image_size, hipMemcpyHostToDevice);
        }

        // Launch kernel to process images in parallel
        int block_size = 256;
        int num_blocks = (num_images * width * height + block_size - 1) / block_size;
        convertToGrayscaleKernelParallel<<<num_blocks, block_size>>>(d_image_data, width, height, num_images);

        // Wait for kernel to finish
        hipDeviceSynchronize();

        // Copy processed image data back to host
        for (int i = 0; i < num_images; ++i)
        {
            hipMemcpy(images[i].getPixelData().data(), d_image_data + i * width * height * 3, image_size, hipMemcpyDeviceToHost);
        }

        // Free device memory
        hipFree(d_image_data);
    }

} // namespace image_library